#include "hip/hip_runtime.h"

/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <string.h>
#include <hipblas.h>
#include <stdint.h>

#define SWEEP_SUCCESS ((void*)1)
#define SWEEP_FAILURE ((void*)0)
#define MAX_DEVICES 256 //256 devices is enough for anyone...

unsigned int testIDs[MAX_DEVICES];
unsigned int testedDevices = 0;
int iterations = 1;
unsigned int speedSetting = 32;
int deviceCount = 0;
pthread_mutex_t lock;
pthread_cond_t condvar;
pthread_t devThreads[MAX_DEVICES];
float elapsedTimes[MAX_DEVICES];

volatile int terminatingDevice = -1;

__global__ void floatMemset(float* ptr, unsigned int length, float value)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    for (; idx < length; idx += stride)
    ptr[idx] = value;
}

void* sgemmSweep(void* devID)
{
    int device = (intptr_t)devID;
    printf("device = %d\n", device);
    float *A, *B, *C, alpha = 1.0, beta = 1.0;
    unsigned int i, j, k;
    if (hipSetDevice(device) != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice(%d) failed\n", device);
        return SWEEP_FAILURE;
    }
    if (cublasInit() != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "Error: cublasInit failed from device %u\n",device);
        return SWEEP_FAILURE;
    }
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    unsigned int iterSize = ((unsigned int)(sqrt((properties.totalGlobalMem-(200*(1<<20)))/24))) & ~(speedSetting-1);
    printf("iterSize = %u\n", iterSize);
    //  printf("Performing %d iterations with increment size %d on device %d...\n", iterations, speedSetting, device);
    for (int curIter = 0; curIter < iterations; curIter++)
    {
        for (i = 128; i < iterSize; i+= speedSetting)
        {
            if (terminatingDevice != -1)
            {
                cublasFree(A);
                cublasFree(B);
                cublasFree(C);
                return SWEEP_SUCCESS;
            }
            printf("Device %d: i = %d\n",device, i);
            float* c_h = (float*)malloc(sizeof(float) * i*i);
            if (!c_h)
            {
                fprintf(stderr, "ERROR: malloc of c_h failed. Aborting.\n");
                terminatingDevice = device;
                return SWEEP_FAILURE;
            }
            if (hipMalloc((void**)&A, i*i*sizeof(float)) != hipSuccess)
            {
                fprintf(stderr, "Error: cublasAlloc(A) failed at i = %d\n", i);
                terminatingDevice = device;
                free(c_h);
                return SWEEP_FAILURE;
            }
            if (hipMalloc((void**)&B, i*i*sizeof(float)) != hipSuccess)
            {
                fprintf(stderr, "Error: cublasAlloc(B) failed at i = %d\n", i);
                terminatingDevice = device;
                free(c_h);
                return SWEEP_FAILURE;
            }

            if (hipMalloc((void**)&C, i*i*sizeof(float)) != hipSuccess)
            {
                fprintf(stderr, "Error: cublasAlloc(C) failed at i = %d\n", i);
                terminatingDevice = device;
                free(c_h);
                return SWEEP_FAILURE;
            }
            floatMemset<<<i/128, 128>>>(A, i*i, 1.0);
            floatMemset<<<i/128, 128>>>(B, i*i, 2.0);
            floatMemset<<<i/128, 128>>>(C, i*i, 3.0);

            if (hipDeviceSynchronize() != hipSuccess)
            {
                fprintf(stderr, "Error: hipDeviceSynchronize returned %s\n", hipGetErrorString(hipGetLastError()));
                terminatingDevice = device;
                free(c_h);
                return SWEEP_FAILURE;
            }
            float result = 2.0 * i + 3.0;

      hipblasSgemm('n', 'n', i, i, i, alpha, A, i, B, i, beta, C, i);
      if (cublasGetError() != HIPBLAS_STATUS_SUCCESS)
      {
          fprintf(stderr, "Error: hipblasSgemm failed!\n");
          terminatingDevice = device;
          free(c_h);
          return SWEEP_FAILURE;
      }
      hipMemcpy(c_h, C, sizeof(float)*i*i, hipMemcpyDeviceToHost);
      for (j = 0; j < i; j++)
      {
          for (k = 0; k < i; k++)
          if (c_h[j*i+k] != result)
          {
              fprintf(stderr, "Error: hipblasSgemm returned an invalid result at location %d,%d in iteration %d on device %d\n", j, k, i, device);
              printf("%f\n", c_h[j*i+k]);
              terminatingDevice = device;
              free(c_h);
              return SWEEP_FAILURE;
          }
      }
      free(c_h);
      cublasFree(A);
      cublasFree(B);
      cublasFree(C);

    }

      //      printf("Finished iteration %d\n", curIter);
      }
      printf("Device %d completed successfully\n", device);
      return SWEEP_SUCCESS;
}

int main (int argc, char** argv)
{
    int i;

    if (argc < 2)
    {
        fprintf(stderr, "usage: %s <speed setting> <iterations>\nSpeed settings:\n0 = iterate by 32 (default)\n1 = iterate by 64\n2 = iterate by 128 (fastest)\n", argv[0]);
        return 1;
    }
    switch (argc)
    {
        case 3:
        sscanf(argv[2], "%d", &iterations);
        case 2:
        unsigned int speed = 0;
        sscanf(argv[1], "%u", &speed);
        if (speed == 2)
        speedSetting = 128;
        else if (speed == 1)
        speedSetting = 64;
    }

    hipGetDeviceCount(&deviceCount);
    printf("deviceCount = %d\n", deviceCount);
    for (i = 0; i < deviceCount; i++)
    {
        struct hipDeviceProp_t properties;
        if (hipGetDeviceProperties(&properties, i) != hipSuccess)
        {
            printf("Could not retrieve properties of device %d\n", i);
            exit(1);
        }

        printf("Testing device %d: %s\n", i, properties.name);
        if ((properties.major != 9999 && properties.minor != 9999)
            //&&
            //((properties.major >= 1 && properties.minor >= 3) ||
            // (properties.major >= 2))
            // && !properties.kernelExecTimeoutEnabled
           )
        {
            testIDs[testedDevices++] = i;
        }
    }
    if (testedDevices == 0)
    {
        printf("No suitable NVIDIA GPUs found. Aborting...\n");
        exit(1);
    }

    for (i = 0; i < testedDevices; i++)
    {
        pthread_create(&devThreads[i], NULL,
                       (sgemmSweep),(void*)((intptr_t)testIDs[i]));
    }
void* returnVal = 0;
for (int i = 0; i < testedDevices; i++)
{
    pthread_join(devThreads[i], &returnVal);
    if (returnVal != SWEEP_SUCCESS)
    {
        printf("ERROR: Failed with device %d. sgemmSweep FAILED.\n", terminatingDevice);
        exit(1);
    }
}
printf("sgemmSweep PASSED.\n");
}

